#include "hip/hip_runtime.h"
#include "Detector.h"
#include <stdlib.h>
#include <memory.h>
#include <assert.h>

#include <vector>
#include <deque>
#include <algorithm>

#include <cv.h>
#include <opencv2\opencv.hpp>
#include <opencv2\gpu\gpu.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <thrust/sort.h>


//for debug
//#include "DebugUtils.h"
//#include <math.h>

using namespace std;
using namespace cv;
using namespace ise;

//declare textures
texture<ushort, 2> texDepth;
texture<float, 2> texSobel;

__constant__ CommonSettings _settingsDev[1];
__constant__ DynamicParameters _dynamicParametersDev[1];
//__constant__ int _floodFillNeighborOffset[6];
__constant__ int _maxHistogramSizeDev[1];

Detector::Detector(const CommonSettings& settings, const cv::Mat& rgbFrame, const cv::Mat& depthFrame, cv::Mat& debugFrame)
    : _settings(settings), _rgbFrame(rgbFrame), _depthFrame(depthFrame), _debugFrame(debugFrame),
    _rgbFrameGpu(settings.rgbHeight, settings.rgbWidth, CV_8UC3),
    _depthFrameGpu(settings.depthHeight, settings.depthWidth, CV_16U),
     _sobelFrameGpu(settings.depthHeight, settings.depthWidth, CV_32F),
    _debugSobelEqFrameGpu(settings.depthHeight, settings.depthWidth, CV_8U),
    _debugFrameGpu(settings.depthHeight, settings.depthWidth, CV_8UC3)
{
	//on device: upload settings to device memory
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_settingsDev), &settings, sizeof(CommonSettings)));

    //init gpu memory for storing strips
    //trips for each row of the depth image are stored in each column of _stripsDev. 
    //The tranpose is to minimize the downloading. 
    //TODO: might destroy coalesced access. What's the tradeoff?
    cudaSafeCall(hipHostMalloc(&_stripsHost, (MAX_STRIPS_PER_ROW + 1) * settings.depthHeight * sizeof(_OmniTouchStripDev)));
    cudaSafeCall(hipMalloc(&_stripsDev, (MAX_STRIPS_PER_ROW + 1) * settings.depthHeight * sizeof(_OmniTouchStripDev)));

    //init memory for storing fingers
    _stripVisitedFlags = new uchar[(MAX_STRIPS_PER_ROW + 1) * settings.depthHeight];

	//init histogram for debug
	_maxHistogramSize = _settings.maxDepthValue * 48 * 2;
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_maxHistogramSizeDev), &_maxHistogramSize, sizeof(int)));
	
	//allocate memory for flood test visited flag
	_floodHitTestVisitedFlag = new uchar[_settings.depthWidth * _settings.depthHeight];
    
	//init vectors
	_fingers.reserve(ISE_MAX_FINGER_NUM);
}

Detector::~Detector()
{
    cudaSafeCall(hipFree(_stripsDev));
    cudaSafeCall(hipHostFree(_stripsHost));
    
    delete [] _stripVisitedFlags;
    delete [] _floodHitTestVisitedFlag;
}

//update the parameters used by the algorithm
void Detector::updateDynamicParameters(const DynamicParameters& parameters)
{
	_parameters = parameters;
	
    //on device: upload parameters to device memory
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_dynamicParametersDev), &parameters, sizeof(DynamicParameters)));
}

//the algorithm goes here. The detection algorithm runs per frame. The input is rgbFrame and depthFrame. The output is the return value, and also the debug frame.
//have a look at main() to learn how to use this.
FingerDetectionResults Detector::detect()
{
	//_iseHistEqualize(depthFrame, debugFrame);

    //_debugFrame.setTo(Scalar(0,0,0));	//set debug frame to black, can also done at GPU
    _debugFrameGpu.setTo(Scalar(0,0,0));

    _depthFrameGpu.upload(_depthFrame);
	sobel();
    
    //bind sobel for following usage
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    gpu::PtrStepSzb ptrStepSz(_sobelFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texSobel, ptrStepSz.data, desc, ptrStepSz.cols, ptrStepSz.rows, ptrStepSz.step));

    //bind depth
    hipChannelFormatDesc descDepth = hipCreateChannelDesc<ushort>();
    gpu::PtrStepSzb ptrStepSzDepth(_depthFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texDepth, ptrStepSzDepth.data, descDepth, ptrStepSzDepth.cols, ptrStepSzDepth.rows, ptrStepSzDepth.step));

    refineDebugImage();
    findStrips();

    //unbind textures
    cudaSafeCall(hipUnbindTexture(texSobel));
    cudaSafeCall(hipUnbindTexture(texDepth));
    
    _debugFrameGpu.download(_debugFrame);

    findFingers();
    floodHitTest();

	FingerDetectionResults r;

	r.error = 0;
	r.fingerCount = _fingers.size() < ISE_MAX_FINGER_NUM ? _fingers.size() : ISE_MAX_FINGER_NUM;
	for (int i = 0; i < r.fingerCount; i++)
	{
		r.fingers[i].tipX = _fingers[i].tipX;
		r.fingers[i].tipY = _fingers[i].tipY;
		r.fingers[i].tipZ = _fingers[i].tipZ;
		r.fingers[i].endX = _fingers[i].endX;
		r.fingers[i].endY = _fingers[i].endY;
		r.fingers[i].endZ = _fingers[i].endZ;
		r.fingers[i].isOnSurface = _fingers[i].isOnSurface ? 1 : 0;
	}

	return r;
}


const ushort* Detector::ushortValAt(const cv::Mat& mat, int row, int col)
{
    assert(mat.type() == CV_16U);
    return (ushort*)(mat.data + row * mat.step + col * sizeof(ushort));
}

float* Detector::floatValAt(cv::Mat& mat, int row, int col)
{
    assert(mat.type() == CV_32F);
    return (float*)(mat.data + row * mat.step + col * sizeof(float));
}

uchar* Detector::rgb888ValAt(cv::Mat& mat, int row, int col)
{
    assert(mat.type() == CV_8UC3);
    return (uchar*)(mat.data + row * mat.step + col * 3);
}

int Detector::divUp(int total, int grain)
{
    return (total + grain - 1) / grain;
}

void Detector::cudaSafeCall(hipError_t err)
{
    //TODO: better handler
    if (err != 0)
    {
        printf("%s\n", hipGetErrorString(err));
        assert(0); 
    }
}

void Detector::convertProjectiveToRealWorld(int x, int y, int depth, double& rx, double& ry, double& rz)
{
	rx = (x / (double)_settings.depthWidth - 0.5) * depth * _settings.kinectIntrinsicParameters.realWorldXToZ;
	ry = (0.5 - y / (double)_settings.depthHeight) * depth * _settings.kinectIntrinsicParameters.realWorldYToZ;
	rz = depth / 100.0 * _settings.kinectIntrinsicParameters.depthSlope + _settings.kinectIntrinsicParameters.depthIntercept;
}

double Detector::getSquaredDistanceInRealWorld(int x1, int y1, int depth1, int x2, int y2, int depth2)
{
	double rx1, ry1, rz1, rx2, ry2, rz2;

	convertProjectiveToRealWorld(x1, y1, depth1, rx1, ry1, rz1);
	convertProjectiveToRealWorld(x2, y2, depth2, rx2, ry2, rz2);

	return ((rx1 - rx2) * (rx1 - rx2) + (ry1 - ry2) * (ry1 - ry2) + (rz1 - rz2) * (rz1 - rz2));
}

void Detector::sobel()
{
    cv::gpu::Sobel(_depthFrameGpu, _sobelFrameGpu, CV_32F, 1, 0, 5, -1);
}

__device__ _FloatPoint3D convertProjectiveToRealWorld(_IntPoint3D p)
{
    _FloatPoint3D r;
    r.x = (p.x / (float)_settingsDev[0].depthWidth - 0.5f) * p.z * _settingsDev[0].kinectIntrinsicParameters.realWorldXToZ;
    r.y = (0.5f - p.y / (float)_settingsDev[0].depthHeight) * p.z * _settingsDev[0].kinectIntrinsicParameters.realWorldYToZ;
    r.z = p.z / 100.0f * _settingsDev[0].kinectIntrinsicParameters.depthSlope + _settingsDev[0].kinectIntrinsicParameters.depthIntercept;

    return r;
}

__device__ float getSquaredDistanceInRealWorld(_IntPoint3D p1, _IntPoint3D p2)
{
    _FloatPoint3D rp1, rp2;

    rp1 = convertProjectiveToRealWorld(p1);
	rp2 = convertProjectiveToRealWorld(p2);

    return ((rp1.x - rp2.x) * (rp1.x - rp2.x) + (rp1.y - rp2.y) * (rp1.y - rp2.y) + (rp1.z - rp2.z) * (rp1.z - rp2.z));
}

__device__ int maxStripRowCountDev;

__global__ void findStripsKernel(gpu::PtrStepb debugPtr, _OmniTouchStripDev* resultPtr)
{
    extern __shared__ int stripCount[];
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    stripCount[threadIdx.x] = 1;

    if (row < _settingsDev[0].depthHeight)
    {
	    StripState state = StripSmooth;
	    int partialMin, partialMax;
	    int partialMinPos, partialMaxPos;

	    for (int col = 0; col < _settingsDev[0].depthWidth; col++)
	    {
		    float currVal = tex2D(texSobel, col, row);
        
        
		    switch(state)
		    {
		    case StripSmooth:	//TODO: smooth
			    if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    partialMax = currVal;
				    partialMaxPos = col;
				    state = StripRising;
			    }
			    break;

		    case StripRising:
			    if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    if (currVal > partialMax)
				    {
					    partialMax = currVal;
					    partialMaxPos = col;
				    }
			    }
			    else 
			    {
				    state = StripMidSmooth;
			    }
			    break;

		    case StripMidSmooth:
			    if (currVal < -_dynamicParametersDev[0].omniTouchParam.fingerFallingThreshold)
			    {
				    partialMin = currVal;
				    partialMinPos = col;
				    state = StripFalling;
			    }
			    else if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    //previous trial faied, start over
				    partialMax = currVal;
				    partialMaxPos = col;
				    state = StripRising;
			    }
			    break;

		    case StripFalling:
			    if (currVal < -_dynamicParametersDev[0].omniTouchParam.fingerFallingThreshold)
			    {
				    if (currVal < partialMin)
				    {
					    partialMin = currVal;
					    partialMinPos = col;
				    }
			    }
			    else
			    {
                    ushort depth = tex2D(texDepth, (partialMaxPos + partialMinPos) / 2, row);
				
                    _IntPoint3D p1, p2;
                    p1.x = partialMaxPos;
                    p1.y = row;
                    p1.z = depth;
                    p2.x = partialMinPos;
                    p2.y = row;
                    p2.z = depth;

				    float distSquared = getSquaredDistanceInRealWorld(p1, p2);

				    if (distSquared >= _dynamicParametersDev[0].omniTouchParam.fingerWidthMin * _dynamicParametersDev[0].omniTouchParam.fingerWidthMin 
					    && distSquared <= _dynamicParametersDev[0].omniTouchParam.fingerWidthMax * _dynamicParametersDev[0].omniTouchParam.fingerWidthMax)
				    {
					    for (int tj = partialMaxPos; tj <= partialMinPos; tj++)
					    {
                            //uchar* pixel = debugPtr.data + row * debugPtr.step + tj * 3;
                            uchar* pixel = debugPtr.ptr(row) + tj * 3;
						    pixel[1] = 255;
					    }

                        int resultOffset = stripCount[threadIdx.x] * _settingsDev[0].depthHeight + row;
                        resultPtr[resultOffset].start = partialMaxPos;
                        resultPtr[resultOffset].end = partialMinPos;
                        resultPtr[resultOffset].row = row;
                        stripCount[threadIdx.x]++;

					    partialMax = currVal;
					    partialMaxPos = col;
				    }

				    state = StripSmooth;
			    }
			    break;
		    } //switch 

            if (stripCount[threadIdx.x] > Detector::MAX_STRIPS_PER_ROW)
            {
                break;
            }
	    } //for 

        //the first row stores count for each column
        //resultPtr[row].start = 1;   //this field unused
        resultPtr[row].end = stripCount[threadIdx.x];
    }   //if row < 0

    __syncthreads();
    //map-recude to find the local maximum strip count
    int total = blockDim.x;
    //int mid = (blockDim.x + 1) / 2;    //div up
    while (total > 1) 
    {
        int mid = (total + 1) / 2;
        if (threadIdx.x < mid)
        {
            if ( (threadIdx.x + mid < total) && stripCount[threadIdx.x + mid] > stripCount[threadIdx.x] ) 
            {
                stripCount[threadIdx.x] = stripCount[threadIdx.x + mid];
            }
        }
        __syncthreads();
        total = mid;
    } 

    if (threadIdx.x == 0)
    {
        atomicMax(&maxStripRowCountDev, stripCount[0]);
    }
}

void Detector::findStrips()
{
    //TODO: what if maximum thread < depthHeight? 
    //the third params: shared memory size in BYTES
    int* maxStripRowCountDevPtr;
    cudaSafeCall(hipGetSymbolAddress((void**)&maxStripRowCountDevPtr, maxStripRowCountDev));
    cudaSafeCall(hipMemset(maxStripRowCountDevPtr, 0, sizeof(int)));

    //turns out 1 block is the best even though profiler suggests more blocks
    int nThread = _settings.depthHeight;    
    int nBlock = 1; //divUp(_settings.depthHeight, nThread);
    findStripsKernel<<<nBlock, nThread, nThread * sizeof(int)>>>(_debugFrameGpu, _stripsDev);
    cudaSafeCall(hipGetLastError());

    cudaSafeCall(hipMemcpyFromSymbol(&_maxStripRowCount, HIP_SYMBOL(maxStripRowCountDev), sizeof(int)));

}

void Detector::findFingers()
{
    //download strips
    //download effective data, there are maxStripCount + 1 rows. The extra row stores count of strips for each column
    cudaSafeCall(hipMemcpy(_stripsHost, _stripsDev, _maxStripRowCount * _settings.depthHeight * sizeof(_OmniTouchStripDev), hipMemcpyDeviceToHost));
    //TODO: according to profiler, this trick seems not necessary. consider optimize for coelesence? 
  
    //init visited flags; 
    memset(_stripVisitedFlags, 0, _settings.depthHeight * _maxStripRowCount);

    //init global finger count
    _fingers.clear();
	
	for (int row = 0; row < _settings.depthHeight; row++)
	{
        for (int col = 0; col < _stripsHost[row].end - 1; col++)
        {
            int stripOffset = (col + 1) * _settings.depthHeight + row;

			if (_stripVisitedFlags[stripOffset] > 0)
			{
				continue;
			}

            _stripBuffer.clear();
            _stripBuffer.push_back(_stripsHost + stripOffset);
            _stripVisitedFlags[stripOffset] = 1;

			//search down
			int blankCounter = 0;
			for (int si = row; si < _settings.depthHeight; si++)   
			{
                _OmniTouchStripDev* currTop = _stripBuffer[_stripBuffer.size() - 1];

				//search strip
				bool stripFound = false;
                
                int searchDownOffset = _settings.depthHeight + si;

                for (int sj = 0; sj < _stripsHost[si].end - 1; ++sj, searchDownOffset += _settings.depthHeight)
				{
					if (_stripVisitedFlags[searchDownOffset])
					{
						continue;
					}

                    _OmniTouchStripDev* candidate = _stripsHost + searchDownOffset;

                    if (candidate->end > currTop->start && candidate->start < currTop->end)	//overlap!
					{
                        _stripBuffer.push_back(_stripsHost + searchDownOffset);
                        
                        //Note: race condition happens here. But won't generate incorrect results.
                        _stripVisitedFlags[searchDownOffset] = 1;
						
                        stripFound = true;
						break;
					}
				}

				if (!stripFound) //blank
				{
					blankCounter++;
					if (blankCounter > _parameters.omniTouchParam.stripMaxBlankPixel)
					{
						//Too much blank, give up
						break;
					}
				}
			}

			//check length
			_OmniTouchStripDev* first = _stripBuffer[0];
            _OmniTouchStripDev* last = _stripBuffer[_stripBuffer.size() - 1];
            
            OmniTouchFinger finger;

            //int firstMidCol = (first->start + first->end) / 2;
            finger.tipX = (first->start + first->end) / 2;
            finger.tipY = first->row;
			//int lastMidCol = (last->start + last->end) / 2;
            finger.endX = (last->start + last->end) / 2;
            finger.endY = last->row;

            finger.tipZ = *(ushort*)(_depthFrame.ptr((first->row + last->row) / 2) + (finger.tipX + finger.endX) / 2 * sizeof(ushort));
            finger.endZ = finger.tipZ;
			
            double lengthSquared = getSquaredDistanceInRealWorld(finger.tipX, finger.tipY, finger.tipZ, finger.endX, finger.endY, finger.endZ);
			int pixelLength = finger.endY - finger.tipY + 1;
			
            if (pixelLength >= _parameters.omniTouchParam.fingerMinPixelLength 
				&& lengthSquared >= _parameters.omniTouchParam.fingerLengthMin * _parameters.omniTouchParam.fingerLengthMin 
				&& lengthSquared <= _parameters.omniTouchParam.fingerLengthMax * _parameters.omniTouchParam.fingerLengthMax)	//finger!
			{
				//fill back
				int bufferPos = -1;
				for (int rowFill = first->row; rowFill <= last->row; rowFill++)
				{
					int leftCol, rightCol;
                    _OmniTouchStripDev* nextBufferItem = _stripBuffer[bufferPos + 1];

					if (rowFill == nextBufferItem->row)	//find next detected row
					{
                        leftCol = nextBufferItem->start;
                        rightCol = nextBufferItem->end;
                        bufferPos++;
					}
					else	//in blank area, interpolate
					{
                        _OmniTouchStripDev* thisBufferItem = _stripBuffer[bufferPos];

						float ratio = (float)(rowFill - thisBufferItem->row) / (float)(nextBufferItem->row - thisBufferItem->row);
                        leftCol = (int)(thisBufferItem->start + (nextBufferItem->start - thisBufferItem->start) * ratio + 0.5f);
                        rightCol = (int)(thisBufferItem->end + (nextBufferItem->end - thisBufferItem->end) * ratio + 0.5f);
					}

					for (int colFill = leftCol; colFill <= rightCol; colFill++)
					{
                        uchar* dstPixel = _debugFrame.ptr(rowFill) + colFill * 3;
                        //uchar* dstPixel = debugPtr.data + rowFill * debugPtr.step + colFill * 3;
						dstPixel[0] = 255;
						dstPixel[2] = 255;
					}
				}

                _fingers.push_back(finger);
			} // check length
		
        }   // for each col
	} //for each row

    sort(_fingers.begin(), _fingers.end());
}


void Detector::floodHitTest()
{
    /*if (_fingerCount > 0)
    {
        //TODO: bad scalability (when image goes large) and too many syncthreads
        //floodHitTestKernel<<<_fingerCount, 512, 512>>>(_debugFrameGpu, _fingersDev);
        floodHitTestKernel<<<_fingerCount, 512, 512 * sizeof(_ShortPoint2D)>>>(_debugFrameGpu, _fingersDev);
        cudaSafeCall(hipGetLastError());
    
        //download result
        cudaSafeCall(hipMemcpy(_fingersHost, _fingersDev, _fingerCount * sizeof(_OmniTouchFingerDev), hipMemcpyDeviceToHost));
    }*/

	static const int neighborOffset[3][2] =
	{
		{-1, 0},
		{1, 0},
		{0, -1}
	};

	for (vector<OmniTouchFinger>::iterator it = _fingers.begin(); it != _fingers.end(); ++it)
	{
		deque<_IntPoint3D> dfsQueue;
		int area = 0;
		memset(_floodHitTestVisitedFlag, 0, _settings.depthWidth * _settings.depthHeight);

		ushort tipDepth = *ushortValAt(_depthFrame, it->tipY, it->tipX);
		_IntPoint3D p;
		p.x = it->tipX;
		p.y = it->tipY;
		p.z = it->tipZ;
		dfsQueue.push_back(p);

		while(!dfsQueue.empty())
		{
			_IntPoint3D centerPoint = dfsQueue.front();
			dfsQueue.pop_front();

			for (int i = 0; i < 3; i++)
			{
				int row = centerPoint.y + neighborOffset[i][1];
				int col = centerPoint.x + neighborOffset[i][0];

				if (row < 0 || row >= _settings.depthHeight || col < 0 || col >= _settings.depthWidth
					|| _floodHitTestVisitedFlag[row * _settings.depthWidth + col] > 0)
				{
					continue;
				}

				ushort neiborDepth = *ushortValAt(_depthFrame, row, col);
				if (abs(neiborDepth - centerPoint.z) > _parameters.omniTouchParam.clickFloodMaxGrad)
				{
					continue;					
				}

				p.x = col;
				p.y = row;
				p.z = neiborDepth;
				dfsQueue.push_back(p);
				area++;
				_floodHitTestVisitedFlag[row * _settings.depthWidth + col] = 255;

				uchar* dstPixel = rgb888ValAt(_debugFrame, row, col);
				dstPixel[0] = 255;
				dstPixel[1] = 255;
				dstPixel[2] = 0;
			}

			if (area >= _parameters.omniTouchParam.clickFloodArea)
			{
				it->isOnSurface = true;
				break;
			}
		}
	}

}

__global__ void convertScaleAbsKernel(gpu::PtrStepb debugSobelEqPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < _settingsDev[0].depthWidth && y < _settingsDev[0].depthHeight)
    {
        float sobel = tex2D(texSobel, x, y);
        uchar res = (uchar)(fabsf(sobel) / (float)(_maxHistogramSizeDev[0]) * 256.0f + 0.5f);
        *(debugSobelEqPtr.ptr(y) + x) = res;
    }
}

__global__ void refineDebugImageKernel(gpu::PtrStepb debugPtr, gpu::PtrStepb sobelEqPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < _settingsDev[0].depthWidth && y < _settingsDev[0].depthHeight)
    {
        uchar* dstPixel = debugPtr.ptr(y) + x * 3;

        if (dstPixel[0] == 255 || dstPixel[1] == 255 || dstPixel[2] == 255)
		{
			//leave as is
		} 
		else
		{
            uchar sobelEq = *(sobelEqPtr.ptr(y) + x);
            float sobelVal = tex2D(texSobel, x, y);

            if (sobelVal >= 0)
            {
                dstPixel[0] = 0;
				dstPixel[2] = sobelEq;
            } else 
            {
                dstPixel[0] = sobelEq;
				dstPixel[2] = 0;
            }
            dstPixel[1] = 0;
		}
    }
}

void Detector::refineDebugImage()
{
    //truncate and eq histogram on sobel
    dim3 threads(16, 32);
    dim3 grid(divUp(_settings.depthWidth, threads.x), divUp(_settings.depthHeight, threads.y));
    convertScaleAbsKernel<<<grid, threads>>>(_debugSobelEqFrameGpu);
    cudaSafeCall(hipGetLastError());

    gpu::equalizeHist(_debugSobelEqFrameGpu, _debugSobelEqFrameGpu);
    
	//draw the image
    refineDebugImageKernel<<<grid, threads>>>(_debugFrameGpu, _debugSobelEqFrameGpu);
    cudaSafeCall(hipGetLastError());
    
}


#include "hip/hip_runtime.h"
#include "Detector.h"
#include <stdlib.h>
#include <memory.h>
#include <assert.h>

#include <vector>
#include <deque>
#include <algorithm>

#include <cv.h>
#include <opencv2\opencv.hpp>
#include <opencv2\gpu\gpu.hpp>
#include <opencv2\gpu\stream_accessor.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
//#include <thrust/sort.h>


//for debug
//#include "DebugUtils.h"
//#include <math.h>

using namespace std;
using namespace cv;
using namespace ise;

//declare textures
texture<ushort, 2> texDepth;
texture<float, 2> texSobel;

__constant__ CommonSettings _settingsDev[1];
__constant__ DynamicParameters _dynamicParametersDev[1];
//__constant__ int _floodFillNeighborOffset[6];
__constant__ int _maxHistogramSizeDev[1];

Detector::Detector(const CommonSettings& settings, const cv::Mat& rgbFrame, const cv::Mat& depthFrame, const cv::Mat& depthToColorCoordFrame, cv::Mat& debugFrame)
    : _settings(settings), _rgbFrame(rgbFrame), _depthFrame(depthFrame), _depthToColorCoordFrame(depthToColorCoordFrame), _debugFrame(debugFrame),
    _rgbFrameGpu(settings.rgbHeight, settings.rgbWidth, CV_8UC3),
    _rgbFloatFrameGpu(settings.rgbHeight, settings.rgbWidth, CV_32FC3),
    _rgbLuvFrameGpu(settings.rgbHeight, settings.rgbWidth, CV_32FC3),
    _rgbPdfFrame(settings.rgbHeight, settings.rgbWidth, CV_32F),
    _rgbPdfFrameGpu(settings.rgbHeight, settings.rgbWidth, CV_32F),
    _depthFrameGpu(settings.depthHeight, settings.depthWidth, CV_16U),
    _sobelFrameGpu(settings.depthHeight, settings.depthWidth, CV_32F),
    _sobelFrameBufferGpu(settings.depthHeight, settings.depthWidth, CV_32F),
    _debugFrameGpu(settings.depthHeight, settings.depthWidth, CV_8UC3),
    _debugSobelEqFrameGpu(settings.depthHeight, settings.depthWidth, CV_8U),
    _debugSobelEqHistGpu(1, 256, CV_32SC1),
    _debugSobelEqBufferGpu(settings.depthHeight, settings.depthWidth, CV_8U)
{
	//on device: upload settings to device memory
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_settingsDev), &settings, sizeof(CommonSettings)));

    //page lock
    gpu::registerPageLocked(_rgbPdfFrame);

    //init gpu memory for storing strips
    //trips for each row of the depth image are stored in each column of _stripsDev. 
    //The tranpose is to minimize the downloading. 
    //TODO: might destroy coalesced access. What's the tradeoff?
    cudaSafeCall(hipHostMalloc(&_stripsHost, (MAX_STRIPS_PER_ROW + 1) * settings.depthHeight * sizeof(_OmniTouchStripDev)));
    cudaSafeCall(hipMalloc(&_stripsDev, (MAX_STRIPS_PER_ROW + 1) * settings.depthHeight * sizeof(_OmniTouchStripDev)));

    //init memory for storing fingers
    _stripVisitedFlags = new uchar[(MAX_STRIPS_PER_ROW + 1) * settings.depthHeight];

	//init histogram for debug
	_maxHistogramSize = _settings.maxDepthValue * 48 * 2;
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_maxHistogramSizeDev), &_maxHistogramSize, sizeof(int)));
	
	//allocate memory for flood test visited flag
	_floodHitTestVisitedFlag = new uchar[_settings.depthWidth * _settings.depthHeight];
    
	//init vectors
	_fingers.reserve(ISE_MAX_FINGER_NUM);
}

Detector::~Detector()
{
    gpu::unregisterPageLocked(_rgbPdfFrame);

    cudaSafeCall(hipFree(_stripsDev));
    cudaSafeCall(hipHostFree(_stripsHost));
    
    delete [] _stripVisitedFlags;
    delete [] _floodHitTestVisitedFlag;
}

//update the parameters used by the algorithm
void Detector::updateDynamicParameters(const DynamicParameters& parameters)
{
	_parameters = parameters;
	
    //on device: upload parameters to device memory
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_dynamicParametersDev), &parameters, sizeof(DynamicParameters)));
}

//the algorithm goes here. The detection algorithm runs per frame. The input is rgbFrame and depthFrame. The output is the return value, and also the debug frame.
//have a look at main() to learn how to use this.
FingerDetectionResults Detector::detect()
{
	//_iseHistEqualize(depthFrame, debugFrame);


    gpuProcess();
    
    /*
    _debugFrameGpu.setTo(Scalar(0,0,0));

    _depthFrameGpu.upload(_depthFrame);
	sobel();
    
    //bind sobel for following usage
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    gpu::PtrStepSzb ptrStepSz(_sobelFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texSobel, ptrStepSz.data, desc, ptrStepSz.cols, ptrStepSz.rows, ptrStepSz.step));

    //bind depth
    hipChannelFormatDesc descDepth = hipCreateChannelDesc<ushort>();
    gpu::PtrStepSzb ptrStepSzDepth(_depthFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texDepth, ptrStepSzDepth.data, descDepth, ptrStepSzDepth.cols, ptrStepSzDepth.rows, ptrStepSzDepth.step));

    refineDebugImage();
    findStrips();

    //unbind textures
    cudaSafeCall(hipUnbindTexture(texSobel));
    cudaSafeCall(hipUnbindTexture(texDepth));
    
    _debugFrameGpu.download(_debugFrame);
    */

    findFingers();
    floodHitTest();


	FingerDetectionResults r;

	r.error = 0;
	r.fingerCount = _fingers.size() < ISE_MAX_FINGER_NUM ? _fingers.size() : ISE_MAX_FINGER_NUM;
	for (int i = 0; i < r.fingerCount; i++)
	{
		r.fingers[i].tipX = _fingers[i].tipX;
		r.fingers[i].tipY = _fingers[i].tipY;
		r.fingers[i].tipZ = _fingers[i].tipZ;
		r.fingers[i].endX = _fingers[i].endX;
		r.fingers[i].endY = _fingers[i].endY;
		r.fingers[i].endZ = _fingers[i].endZ;
		r.fingers[i].isOnSurface = _fingers[i].isOnSurface ? 1 : 0;
	}

	return r;
}


const ushort* Detector::ushortValAt(const cv::Mat& mat, int row, int col)
{
    assert(mat.type() == CV_16U);
    return (ushort*)(mat.data + row * mat.step + col * sizeof(ushort));
}

float* Detector::floatValAt(cv::Mat& mat, int row, int col)
{
    assert(mat.type() == CV_32F);
    return (float*)(mat.data + row * mat.step + col * sizeof(float));
}

uchar* Detector::rgb888ValAt(cv::Mat& mat, int row, int col)
{
    assert(mat.type() == CV_8UC3);
    return (uchar*)(mat.data + row * mat.step + col * 3);
}

int Detector::divUp(int total, int grain)
{
    return (total + grain - 1) / grain;
}

void Detector::cudaSafeCall(hipError_t err)
{
    //TODO: better handler
    if (err != 0)
    {
        printf("%s\n", hipGetErrorString(err));
        assert(0); 
    }
}

void Detector::convertProjectiveToRealWorld(int x, int y, int depth, double& rx, double& ry, double& rz)
{
	rx = (x / (double)_settings.depthWidth - 0.5) * depth * _settings.kinectIntrinsicParameters.realWorldXToZ;
	ry = (0.5 - y / (double)_settings.depthHeight) * depth * _settings.kinectIntrinsicParameters.realWorldYToZ;
	rz = depth / 100.0 * _settings.kinectIntrinsicParameters.depthSlope + _settings.kinectIntrinsicParameters.depthIntercept;
}

double Detector::getSquaredDistanceInRealWorld(int x1, int y1, int depth1, int x2, int y2, int depth2)
{
	double rx1, ry1, rz1, rx2, ry2, rz2;

	convertProjectiveToRealWorld(x1, y1, depth1, rx1, ry1, rz1);
	convertProjectiveToRealWorld(x2, y2, depth2, rx2, ry2, rz2);

	return ((rx1 - rx2) * (rx1 - rx2) + (ry1 - ry2) * (ry1 - ry2) + (rz1 - rz2) * (rz1 - rz2));
}

__device__ _FloatPoint3D convertProjectiveToRealWorld(_IntPoint3D p)
{
    _FloatPoint3D r;
    r.x = (p.x / (float)_settingsDev[0].depthWidth - 0.5f) * p.z * _settingsDev[0].kinectIntrinsicParameters.realWorldXToZ;
    r.y = (0.5f - p.y / (float)_settingsDev[0].depthHeight) * p.z * _settingsDev[0].kinectIntrinsicParameters.realWorldYToZ;
    r.z = p.z / 100.0f * _settingsDev[0].kinectIntrinsicParameters.depthSlope + _settingsDev[0].kinectIntrinsicParameters.depthIntercept;

    return r;
}

__device__ float getSquaredDistanceInRealWorld(_IntPoint3D p1, _IntPoint3D p2)
{
    _FloatPoint3D rp1, rp2;

    rp1 = convertProjectiveToRealWorld(p1);
	rp2 = convertProjectiveToRealWorld(p2);

    return ((rp1.x - rp2.x) * (rp1.x - rp2.x) + (rp1.y - rp2.y) * (rp1.y - rp2.y) + (rp1.z - rp2.z) * (rp1.z - rp2.z));
}

__device__ int maxStripRowCountDev;

__global__ void findStripsKernel(gpu::PtrStepb debugPtr, _OmniTouchStripDev* resultPtr)
{
    extern __shared__ int stripCount[];
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    stripCount[threadIdx.x] = 1;

    if (row < _settingsDev[0].depthHeight)
    {
	    StripState state = StripSmooth;
	    int partialMin, partialMax;
	    int partialMinPos, partialMaxPos;

	    for (int col = 0; col < _settingsDev[0].depthWidth; col++)
	    {
		    float currVal = tex2D(texSobel, col, row);
        
        
		    switch(state)
		    {
		    case StripSmooth:	//TODO: smooth
			    if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    partialMax = currVal;
				    partialMaxPos = col;
				    state = StripRising;
			    }
			    break;

		    case StripRising:
			    if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    if (currVal > partialMax)
				    {
					    partialMax = currVal;
					    partialMaxPos = col;
				    }
			    }
			    else 
			    {
				    state = StripMidSmooth;
			    }
			    break;

		    case StripMidSmooth:
			    if (currVal < -_dynamicParametersDev[0].omniTouchParam.fingerFallingThreshold)
			    {
				    partialMin = currVal;
				    partialMinPos = col;
				    state = StripFalling;
			    }
			    else if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    //previous trial faied, start over
				    partialMax = currVal;
				    partialMaxPos = col;
				    state = StripRising;
			    }
			    break;

		    case StripFalling:
			    if (currVal < -_dynamicParametersDev[0].omniTouchParam.fingerFallingThreshold)
			    {
				    if (currVal < partialMin)
				    {
					    partialMin = currVal;
					    partialMinPos = col;
				    }
			    }
			    else
			    {
                    ushort depth = tex2D(texDepth, (partialMaxPos + partialMinPos) / 2, row);
				
                    _IntPoint3D p1, p2;
                    p1.x = partialMaxPos;
                    p1.y = row;
                    p1.z = depth;
                    p2.x = partialMinPos;
                    p2.y = row;
                    p2.z = depth;

				    float distSquared = getSquaredDistanceInRealWorld(p1, p2);

				    if (distSquared >= _dynamicParametersDev[0].omniTouchParam.fingerWidthMin * _dynamicParametersDev[0].omniTouchParam.fingerWidthMin 
					    && distSquared <= _dynamicParametersDev[0].omniTouchParam.fingerWidthMax * _dynamicParametersDev[0].omniTouchParam.fingerWidthMax)
				    {
					    for (int tj = partialMaxPos; tj <= partialMinPos; tj++)
					    {
                            //uchar* pixel = debugPtr.data + row * debugPtr.step + tj * 3;
                            uchar* pixel = debugPtr.ptr(row) + tj * 3;
						    pixel[1] = 255;
					    }

                        int resultOffset = stripCount[threadIdx.x] * _settingsDev[0].depthHeight + row;
                        resultPtr[resultOffset].start = partialMaxPos;
                        resultPtr[resultOffset].end = partialMinPos;
                        resultPtr[resultOffset].row = row;
                        stripCount[threadIdx.x]++;

					    partialMax = currVal;
					    partialMaxPos = col;
				    }

				    state = StripSmooth;
			    }
			    break;
		    } //switch 

            if (stripCount[threadIdx.x] > Detector::MAX_STRIPS_PER_ROW)
            {
                break;
            }
	    } //for 

        //the first row stores count for each column
        //resultPtr[row].start = 1;   //this field unused
        resultPtr[row].end = stripCount[threadIdx.x];
    }   //if row < 0

    __syncthreads();
    //map-recude to find the local maximum strip count
    int total = blockDim.x;
    //int mid = (blockDim.x + 1) / 2;    //div up
    while (total > 1) 
    {
        int mid = (total + 1) / 2;
        if (threadIdx.x < mid)
        {
            if ( (threadIdx.x + mid < total) && stripCount[threadIdx.x + mid] > stripCount[threadIdx.x] ) 
            {
                stripCount[threadIdx.x] = stripCount[threadIdx.x + mid];
            }
        }
        __syncthreads();
        total = mid;
    } 

    if (threadIdx.x == 0)
    {
        atomicMax(&maxStripRowCountDev, stripCount[0]);
    }
}

void Detector::findFingers()
{
    //init visited flags; 
    memset(_stripVisitedFlags, 0, _settings.depthHeight * _maxStripRowCount);

    //init global finger count
    _fingers.clear();
	
	for (int row = 0; row < _settings.depthHeight; row++)
	{
        for (int col = 0; col < _stripsHost[row].end - 1; col++)
        {
            int stripOffset = (col + 1) * _settings.depthHeight + row;

			if (_stripVisitedFlags[stripOffset] > 0)
			{
				continue;
			}

            _stripBuffer.clear();
            _stripBuffer.push_back(_stripsHost + stripOffset);
            _stripVisitedFlags[stripOffset] = 1;

			//search down
			int blankCounter = 0;
			for (int si = row; si < _settings.depthHeight; si++)   
			{
                _OmniTouchStripDev* currTop = _stripBuffer[_stripBuffer.size() - 1];

				//search strip
				bool stripFound = false;
                
                int searchDownOffset = _settings.depthHeight + si;

                for (int sj = 0; sj < _stripsHost[si].end - 1; ++sj, searchDownOffset += _settings.depthHeight)
				{
					if (_stripVisitedFlags[searchDownOffset])
					{
						continue;
					}

                    _OmniTouchStripDev* candidate = _stripsHost + searchDownOffset;

                    if (candidate->end > currTop->start && candidate->start < currTop->end)	//overlap!
					{
                        _stripBuffer.push_back(_stripsHost + searchDownOffset);
                        
                        //Note: race condition happens here. But won't generate incorrect results.
                        _stripVisitedFlags[searchDownOffset] = 1;
						
                        stripFound = true;
						break;
					}
				}

				if (!stripFound) //blank
				{
					blankCounter++;
					if (blankCounter > _parameters.omniTouchParam.stripMaxBlankPixel)
					{
						//Too much blank, give up
						break;
					}
				}
			}

			//check length
			_OmniTouchStripDev* first = _stripBuffer[0];
            _OmniTouchStripDev* last = _stripBuffer[_stripBuffer.size() - 1];
            
            OmniTouchFinger finger;

            //int firstMidCol = (first->start + first->end) / 2;
            finger.tipX = (first->start + first->end) / 2;
            finger.tipY = first->row;
			//int lastMidCol = (last->start + last->end) / 2;
            finger.endX = (last->start + last->end) / 2;
            finger.endY = last->row;

            finger.tipZ = *(ushort*)(_depthFrame.ptr((first->row + last->row) / 2) + (finger.tipX + finger.endX) / 2 * sizeof(ushort));
            finger.endZ = finger.tipZ;
			
            double lengthSquared = getSquaredDistanceInRealWorld(finger.tipX, finger.tipY, finger.tipZ, finger.endX, finger.endY, finger.endZ);
			int pixelLength = finger.endY - finger.tipY + 1;
			
            if (pixelLength >= _parameters.omniTouchParam.fingerMinPixelLength 
				&& lengthSquared >= _parameters.omniTouchParam.fingerLengthMin * _parameters.omniTouchParam.fingerLengthMin 
				&& lengthSquared <= _parameters.omniTouchParam.fingerLengthMax * _parameters.omniTouchParam.fingerLengthMax)	//finger!
			{
				//fill back
				int bufferPos = -1;
				for (int rowFill = first->row; rowFill <= last->row; rowFill++)
				{
					int leftCol, rightCol;
                    _OmniTouchStripDev* nextBufferItem = _stripBuffer[bufferPos + 1];

					if (rowFill == nextBufferItem->row)	//find next detected row
					{
                        leftCol = nextBufferItem->start;
                        rightCol = nextBufferItem->end;
                        bufferPos++;
					}
					else	//in blank area, interpolate
					{
                        _OmniTouchStripDev* thisBufferItem = _stripBuffer[bufferPos];

						float ratio = (float)(rowFill - thisBufferItem->row) / (float)(nextBufferItem->row - thisBufferItem->row);
                        leftCol = (int)(thisBufferItem->start + (nextBufferItem->start - thisBufferItem->start) * ratio + 0.5f);
                        rightCol = (int)(thisBufferItem->end + (nextBufferItem->end - thisBufferItem->end) * ratio + 0.5f);
					}

					for (int colFill = leftCol; colFill <= rightCol; colFill++)
					{
                        uchar* dstPixel = _debugFrame.ptr(rowFill) + colFill * 3;
                        
						dstPixel[0] = 255;
						dstPixel[2] = 255;

                        //read color
                        /*const int* mapCoord = (int*)_depthToColorCoordFrame.ptr(rowFill) + colFill * 2;
                        int cx = mapCoord[0];
                        int cy = mapCoord[1];
                        const uchar* rgbPixel = _rgbFrame.ptr(cy) + cx * 3;
                        //const uchar* rgbPixel = _rgbFrame.ptr(rowFill) + colFill * 3;

                        memcpy(dstPixel, rgbPixel, 3);*/
					}
				}

                _fingers.push_back(finger);
			} // check length
		
        }   // for each col
	} //for each row

    sort(_fingers.begin(), _fingers.end());
}


void Detector::floodHitTest()
{
    /*if (_fingerCount > 0)
    {
        //TODO: bad scalability (when image goes large) and too many syncthreads
        //floodHitTestKernel<<<_fingerCount, 512, 512>>>(_debugFrameGpu, _fingersDev);
        floodHitTestKernel<<<_fingerCount, 512, 512 * sizeof(_ShortPoint2D)>>>(_debugFrameGpu, _fingersDev);
        cudaSafeCall(hipGetLastError());
    
        //download result
        cudaSafeCall(hipMemcpy(_fingersHost, _fingersDev, _fingerCount * sizeof(_OmniTouchFingerDev), hipMemcpyDeviceToHost));
    }*/

	static const int neighborOffset[3][2] =
	{
		{-1, 0},
		{1, 0},
		{0, -1}
	};

	for (vector<OmniTouchFinger>::iterator it = _fingers.begin(); it != _fingers.end(); ++it)
	{
		deque<_IntPoint3D> dfsQueue;
		int area = 0;
		memset(_floodHitTestVisitedFlag, 0, _settings.depthWidth * _settings.depthHeight);

		ushort tipDepth = *ushortValAt(_depthFrame, it->tipY, it->tipX);
		_IntPoint3D p;
		p.x = it->tipX;
		p.y = it->tipY;
		p.z = it->tipZ;
		dfsQueue.push_back(p);

		while(!dfsQueue.empty())
		{
			_IntPoint3D centerPoint = dfsQueue.front();
			dfsQueue.pop_front();

			for (int i = 0; i < 3; i++)
			{
				int row = centerPoint.y + neighborOffset[i][1];
				int col = centerPoint.x + neighborOffset[i][0];

				if (row < 0 || row >= _settings.depthHeight || col < 0 || col >= _settings.depthWidth
					|| _floodHitTestVisitedFlag[row * _settings.depthWidth + col] > 0)
				{
					continue;
				}

				ushort neiborDepth = *ushortValAt(_depthFrame, row, col);
				if (abs(neiborDepth - centerPoint.z) > _parameters.omniTouchParam.clickFloodMaxGrad)
				{
					continue;					
				}

				p.x = col;
				p.y = row;
				p.z = neiborDepth;
				dfsQueue.push_back(p);
				area++;
				_floodHitTestVisitedFlag[row * _settings.depthWidth + col] = 255;

				uchar* dstPixel = rgb888ValAt(_debugFrame, row, col);
				dstPixel[0] = 255;
				dstPixel[1] = 255;
				dstPixel[2] = 0;
			}

			if (area >= _parameters.omniTouchParam.clickFloodArea)
			{
				it->isOnSurface = true;
				break;
			}
		}
	}

}

__global__ void convertScaleAbsKernel(gpu::PtrStepb debugSobelEqPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < _settingsDev[0].depthWidth && y < _settingsDev[0].depthHeight)
    {
        float sobel = tex2D(texSobel, x, y);
        uchar res = (uchar)(fabsf(sobel) / (float)(_maxHistogramSizeDev[0]) * 255.0f + 0.5f);
        *(debugSobelEqPtr.ptr(y) + x) = res;
    }
}

__global__ void refineDebugImageKernel(gpu::PtrStepb debugPtr, gpu::PtrStepb sobelEqPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < _settingsDev[0].depthWidth && y < _settingsDev[0].depthHeight)
    {
        uchar* dstPixel = debugPtr.ptr(y) + x * 3;

        if (dstPixel[0] == 255 || dstPixel[1] == 255 || dstPixel[2] == 255)
		{
			//leave as is
		} 
		else
		{
            uchar sobelEq = *(sobelEqPtr.ptr(y) + x);
            float sobelVal = tex2D(texSobel, x, y);

            if (sobelVal >= 0)
            {
                dstPixel[0] = 0;
				dstPixel[2] = sobelEq;
            } else 
            {
                dstPixel[0] = sobelEq;
				dstPixel[2] = 0;
            }
            dstPixel[1] = 0;
		}
    }
}


__global__ void applySkinColorModel(gpu::PtrStepf luvPtr, gpu::PtrStepf pdfPtr)
{
    const int nComp = 3;
    const float mu[nComp][2] = { {11.2025f, 8.2296f},
                         {35.5613f, 30.1054f},
                         {22.7229f, 19.4680f} };
    const float conv[nComp][2] = { {37.691f, 43.929f},
                            {225.23f, 242.08f},
                            {54.738f, 61.146f} };
    const float prop[nComp] = {0.28847f, 0.24641f, 0.46512f};

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < _settingsDev[0].rgbWidth && y < _settingsDev[0].rgbHeight)
    {
        float* luv = luvPtr.ptr(y) + x * 3;
        float u = luv[1];
        float v = luv[2];

        float p = 0;
        
        #pragma unroll
        for (int i = 0; i < nComp; i++)
        {
            float d = 1.f / (2 * HIP_PI_F * sqrt(conv[i][0] * conv[i][1]));
            float e = expf(-0.5f * (powf(u - mu[i][0], 2) / conv[i][0] + powf(v - mu[i][1], 2) / conv[i][1]));
            p += prop[i] * d * e;
        }

        float* dst = pdfPtr.ptr(y) + x;
        //*dst = p * 500.f;
        //*dst = luv[0] / 100.0f;
        *dst = p * 1000.f;
    }
}


/*__global__ void applySkinColorModel(gpu::PtrStepf luvPtr, gpu::PtrStepf pdfPtr)
{
    const int nComp = 3;
    const double mu[nComp][2] = {{11.2024686283253,          8.22956679034156},
                                {35.5612918973632,          30.1054062096261},
                                {22.7229203550896,          19.4680134168419}};
    const double conv[nComp][2] = { { 37.6914561020612,          43.9294235115161},
                                    { 225.22898713039,          242.081315811438},
                                    { 54.7381847499349,          61.1464611751676} };
    const double prop[nComp] = {0.28846824006064,         0.246407551490279,          0.46512420844908};

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < _settingsDev[0].rgbWidth && y < _settingsDev[0].rgbHeight)
    {
        float* luv = luvPtr.ptr(y) + x * 3;
        double u = luv[1];
        double v = luv[2];

        double p = 0;
        
        #pragma unroll
        for (int i = 0; i < nComp; i++)
        {
            double d = 1.0 / (2.0 * HIP_PI_F * sqrt(conv[i][0] * conv[i][1]));
            double e = exp(-0.5 * (pow(u - mu[i][0], 2) / conv[i][0] + pow(v - mu[i][1], 2) / conv[i][1]));
            p += prop[i] * d * e;
        }

        float* dst = pdfPtr.ptr(y) + x;
        //*dst = p * 500.f;
        //*dst = luv[0] / 100.0f;
        *dst = (float)(p * 1000.0);
    }
}*/

void Detector::gpuProcess()
{
    hipStream_t cudaStreamDepthDebug = gpu::StreamAccessor::getStream(_gpuStreamDepthDebug);
    hipStream_t cudaStreamDepthWorking = gpu::StreamAccessor::getStream(_gpuStreamDepthWorking);
    hipStream_t cudaStreamRgbWorking = gpu::StreamAccessor::getStream(_gpuStreamRgbWorking);
    
    _depthFrameGpu.upload(_depthFrame);
    //_gpuStreamDepthWorking.enqueueUpload(_depthFrame, _depthFrameGpu);
    
    //Looks like when running Sobel async, visual profiler won't generate any timeline.
    cv::gpu::Sobel(_depthFrameGpu, _sobelFrameGpu, CV_32F, 1, 0, _sobelFrameBufferGpu, 5, -1);
    //cv::gpu::Sobel(_depthFrameGpu, _sobelFrameGpu, CV_32F, 1, 0, _sobelFrameBufferGpu, 5, -1.0f, BORDER_DEFAULT, -1, _gpuStreamDepthWorking);
    //_gpuStreamDepthWorking.waitForCompletion();

    _gpuStreamDepthWorking.enqueueMemSet(_debugFrameGpu, Scalar(0,0,0));
    
    //bind sobel for following usage
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    gpu::PtrStepSzb ptrStepSz(_sobelFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texSobel, ptrStepSz.data, desc, ptrStepSz.cols, ptrStepSz.rows, ptrStepSz.step));

    //bind depth
    hipChannelFormatDesc descDepth = hipCreateChannelDesc<ushort>();
    gpu::PtrStepSzb ptrStepSzDepth(_depthFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texDepth, ptrStepSzDepth.data, descDepth, ptrStepSzDepth.cols, ptrStepSzDepth.rows, ptrStepSzDepth.step));
    
    //find strips on stream2: upload data
    //TODO: what if maximum thread < depthHeight? 
    //the third params: shared memory size in BYTES
    int* maxStripRowCountDevPtr;
    cudaSafeCall(hipGetSymbolAddress((void**)&maxStripRowCountDevPtr, maxStripRowCountDev));
    cudaSafeCall(hipMemsetAsync(maxStripRowCountDevPtr, 0, sizeof(int), cudaStreamDepthWorking));

    //find strips on stream 2: kernel call
    //turns out 1 block is the best even though profiler suggests more blocks
    int nThread = _settings.depthHeight;    
    int nBlock = 1; //divUp(_settings.depthHeight, nThread);
    findStripsKernel<<<nBlock, nThread, nThread * sizeof(int), cudaStreamDepthWorking>>>(_debugFrameGpu, _stripsDev);
    //cudaSafeCall(hipGetLastError());

    //rgb manipulation
    _gpuStreamRgbWorking.enqueueUpload(_rgbFrame, _rgbFrameGpu);
    _gpuStreamRgbWorking.enqueueConvert(_rgbFrameGpu, _rgbFloatFrameGpu, CV_32FC3, 1.f / 255.f);
    gpu::cvtColor(_rgbFloatFrameGpu, _rgbLuvFrameGpu, CV_RGB2Luv, 0, _gpuStreamRgbWorking);

    //rgb skin color model
    dim3 rgbThreads(16, 32);
    dim3 rgbGrid(divUp(_settings.rgbWidth, rgbThreads.x), divUp(_settings.rgbHeight, rgbThreads.y));
    applySkinColorModel<<<rgbGrid, rgbThreads, 0, cudaStreamRgbWorking>>>(_rgbLuvFrameGpu, _rgbPdfFrameGpu);
    cudaSafeCall(hipGetLastError());

    //refine debug image on stream1: kernel call
    dim3 threads(16, 32);
    dim3 grid(divUp(_settings.depthWidth, threads.x), divUp(_settings.depthHeight, threads.y));
    convertScaleAbsKernel<<<grid, threads, 0, cudaStreamDepthDebug>>>(_debugSobelEqFrameGpu);
    //cudaSafeCall(hipGetLastError());

    gpu::equalizeHist(_debugSobelEqFrameGpu, _debugSobelEqFrameGpu, _debugSobelEqHistGpu, _debugSobelEqBufferGpu, _gpuStreamDepthDebug);
    
    //rgb download
    _gpuStreamRgbWorking.enqueueDownload(_rgbPdfFrameGpu, _rgbPdfFrame);

    //find strips on stream 2: download data
    cudaSafeCall(hipMemcpyFromSymbolAsync(&_maxStripRowCount, HIP_SYMBOL(maxStripRowCountDev), sizeof(int), 0, hipMemcpyDeviceToHost, cudaStreamDepthWorking));

    //download strips
    //download effective data, there are maxStripCount + 1 rows. The extra row stores count of strips for each column
    cudaSafeCall(hipMemcpyAsync(_stripsHost, _stripsDev, _maxStripRowCount * _settings.depthHeight * sizeof(_OmniTouchStripDev), 
        hipMemcpyDeviceToHost, cudaStreamDepthWorking));
    //TODO: according to profiler, this trick seems not necessary. consider optimize for coelesence? 
  
    _gpuStreamDepthDebug.waitForCompletion();
    _gpuStreamDepthWorking.waitForCompletion();  
    cudaSafeCall(hipGetLastError());

    //draw the debug image
    refineDebugImageKernel<<<grid, threads, 0, cudaStreamDepthDebug>>>(_debugFrameGpu, _debugSobelEqFrameGpu);
    cudaSafeCall(hipGetLastError());
    
    //_debugFrameGpu.download(_debugFrame);
    _gpuStreamDepthDebug.enqueueDownload(_debugFrameGpu, _debugFrame);
    _gpuStreamDepthDebug.waitForCompletion();


    //unbind textures
    cudaSafeCall(hipUnbindTexture(texSobel));
    cudaSafeCall(hipUnbindTexture(texDepth));
    

    _gpuStreamRgbWorking.waitForCompletion();
    
}
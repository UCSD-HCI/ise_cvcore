#include "hip/hip_runtime.h"
#include "Detector.h"
#include <stdlib.h>
#include <memory.h>
#include <assert.h>

#include <vector>
#include <deque>
#include <algorithm>

#include <cv.h>
#include <opencv2\opencv.hpp>
#include <opencv2\gpu\gpu.hpp>
#include <opencv2\gpu\stream_accessor.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

using namespace std;
using namespace cv;
using namespace ise;

//declare textures
texture<ushort, 2> texDepth;
texture<float, 2> texSobel;
texture<ushort, 2> texTrDepth;
texture<float, 2> texTrSobel;

__constant__ CommonSettings _settingsDev[1];
__constant__ DynamicParameters _dynamicParametersDev[1];
//__constant__ int _floodFillNeighborOffset[6];
__constant__ int _maxHistogramSizeDev[1];

template <_ImageDirection dir>
__device__ int depthWidth()
{
    return (dir == DirTransposed ? _settingsDev[0].depthHeight : _settingsDev[0].depthWidth);
}

template <_ImageDirection dir>
__device__ int depthHeight()
{
    return (dir == DirTransposed ? _settingsDev[0].depthWidth : _settingsDev[0].depthHeight);
}

//choose texture according to direction
#define dirTex2D(dir, tex, texTr, x, y) ((dir) == DirTransposed ? tex2D((texTr), (x), (y)) : tex2D((tex), (x), (y)))

void Detector::cudaSafeCall(hipError_t err)
{
    //TODO: better handler
    if (err != 0)
    {
        const char* errStr = hipGetErrorString(err);
        printf("%s\n", errStr);
        assert(0); 
    }
}

void Detector::cudaInit()
{
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_settingsDev), &_settings, sizeof(CommonSettings)));

    //init gpu memory for storing strips
    //trips for each row of the depth image are stored in each column of _stripsDev. 
    //The tranpose is to minimize the downloading. 
    //TODO: might destroy coalesced access. What's the tradeoff?
    cudaSafeCall(hipHostMalloc(&_stripsHost, (MAX_STRIPS_PER_ROW + 1) * _settings.depthHeight * sizeof(_OmniTouchStripDev)));
    cudaSafeCall(hipMalloc(&_stripsDev, (MAX_STRIPS_PER_ROW + 1) * _settings.depthHeight * sizeof(_OmniTouchStripDev)));

    //strips transposed
    cudaSafeCall(hipHostMalloc(&_transposedStripsHost, (MAX_STRIPS_PER_ROW + 1) * _settings.depthWidth * sizeof(_OmniTouchStripDev)));
    cudaSafeCall(hipMalloc(&_transposedStripsDev, (MAX_STRIPS_PER_ROW + 1) * _settings.depthWidth * sizeof(_OmniTouchStripDev)));


    //init histogram for debug
	_maxHistogramSize = _settings.maxDepthValue * 48 * 2;
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_maxHistogramSizeDev), &_maxHistogramSize, sizeof(int)));
	
}

void Detector::cudaRelease()
{
    cudaSafeCall(hipFree(_stripsDev));
    cudaSafeCall(hipHostFree(_stripsHost));

    cudaSafeCall(hipFree(_transposedStripsDev));
    cudaSafeCall(hipHostFree(_transposedStripsHost));
}

//update the parameters used by the algorithm
void Detector::updateDynamicParameters(const DynamicParameters& parameters)
{
	_parameters = parameters;
	
    //on device: upload parameters to device memory
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_dynamicParametersDev), &parameters, sizeof(DynamicParameters)));
}

__device__ _FloatPoint3D convertProjectiveToRealWorld(_IntPoint3D p)
{
    _FloatPoint3D r;
    r.x = (p.x / (float)_settingsDev[0].depthWidth - 0.5f) * p.z * _settingsDev[0].kinectIntrinsicParameters.realWorldXToZ;
    r.y = (0.5f - p.y / (float)_settingsDev[0].depthHeight) * p.z * _settingsDev[0].kinectIntrinsicParameters.realWorldYToZ;
    r.z = p.z / 100.0f * _settingsDev[0].kinectIntrinsicParameters.depthSlope + _settingsDev[0].kinectIntrinsicParameters.depthIntercept;

    return r;
}

__device__ float getSquaredDistanceInRealWorld(_IntPoint3D p1, _IntPoint3D p2)
{
    _FloatPoint3D rp1, rp2;

    rp1 = convertProjectiveToRealWorld(p1);
	rp2 = convertProjectiveToRealWorld(p2);

    return ((rp1.x - rp2.x) * (rp1.x - rp2.x) + (rp1.y - rp2.y) * (rp1.y - rp2.y) + (rp1.z - rp2.z) * (rp1.z - rp2.z));
}

__device__ int maxStripRowCountDev;
__device__ int trMaxStripRowCountDev;

template <_ImageDirection dir>
__global__ void findStripsKernel(gpu::PtrStepb debugPtr, _OmniTouchStripDev* resultPtr)
{
    extern __shared__ int stripCount[];
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    stripCount[threadIdx.x] = 1;

    int width = depthWidth<dir>();
    int height = depthHeight<dir>();

    if (row < height)
    {
	    StripState state = StripSmooth;
	    int partialMin, partialMax;
	    int partialMinPos, partialMaxPos;

	    for (int col = 0; col < width; col++)
	    {
		    float currVal = dirTex2D(dir, texSobel, texTrSobel, col, row);
            ushort depthVal = dirTex2D(dir, texDepth, texTrDepth, col, row);
        
		    switch(state)
		    {
		    case StripSmooth:	//TODO: smooth
                if (depthVal == 0 || depthVal == Detector::DEPTH_UNKNOWN_VALUE)
                {
                    //same state
                }
			    else if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    partialMax = currVal;
				    partialMaxPos = col;
				    state = StripRising;
			    }
			    break;

		    case StripRising:
                if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    if (currVal > partialMax)
				    {
					    partialMax = currVal;
					    partialMaxPos = col;
				    }
			    }
			    else 
			    {
				    state = StripMidSmooth;
			    }
			    break;

		    case StripMidSmooth:
                if (currVal < -_dynamicParametersDev[0].omniTouchParam.fingerFallingThreshold)
			    {
				    partialMin = currVal;
				    partialMinPos = col;
				    state = StripFalling;
			    }
			    else if (currVal > _dynamicParametersDev[0].omniTouchParam.fingerRisingThreshold)
			    {
				    //previous trial faied, start over
				    partialMax = currVal;
				    partialMaxPos = col;
				    state = StripRising;
			    }
			    break;

		    case StripFalling:
			    if (depthVal != 0 && depthVal != Detector::DEPTH_UNKNOWN_VALUE 
                    && currVal < -_dynamicParametersDev[0].omniTouchParam.fingerFallingThreshold)
			    {
				    if (currVal < partialMin)
				    {
					    partialMin = currVal;
					    partialMinPos = col;
				    }
			    }
			    else
			    {
                    ushort depth = dirTex2D(dir, texDepth, texTrDepth, (partialMaxPos + partialMinPos) / 2, row);
				
                    _IntPoint3D p1, p2;
                    p1.x = partialMaxPos;
                    p1.y = row;
                    p1.z = depth;
                    p2.x = partialMinPos;
                    p2.y = row;
                    p2.z = depth;

				    float distSquared = getSquaredDistanceInRealWorld(p1, p2);

				    if (distSquared >= _dynamicParametersDev[0].omniTouchParam.fingerWidthMin * _dynamicParametersDev[0].omniTouchParam.fingerWidthMin 
					    && distSquared <= _dynamicParametersDev[0].omniTouchParam.fingerWidthMax * _dynamicParametersDev[0].omniTouchParam.fingerWidthMax)
				    {
					    for (int tj = partialMaxPos; tj <= partialMinPos; tj++)
					    {
                            //uchar* pixel = debugPtr.data + row * debugPtr.step + tj * 3;
                            uchar* pixel = debugPtr.ptr(row) + tj * 3;
						    pixel[1] = 255;
					    }

                        int resultOffset = stripCount[threadIdx.x] * height + row;
                        resultPtr[resultOffset].start = partialMaxPos;
                        resultPtr[resultOffset].end = partialMinPos;
                        resultPtr[resultOffset].row = row;
                        stripCount[threadIdx.x]++;

					    partialMax = currVal;
					    partialMaxPos = col;
				    }

				    state = StripSmooth;
			    }
			    break;
		    } //switch 

            if (stripCount[threadIdx.x] > Detector::MAX_STRIPS_PER_ROW)
            {
                break;
            }
	    } //for 

        //the first row stores count for each column
        //resultPtr[row].start = 1;   //this field unused
        resultPtr[row].end = stripCount[threadIdx.x];
    }   //if row < 0

    __syncthreads();
    //map-recude to find the local maximum strip count
    int total = blockDim.x;
    while (total > 1) 
    {
        int mid = (total + 1) / 2;
        if (threadIdx.x < mid)
        {
            if ( (threadIdx.x + mid < total) && stripCount[threadIdx.x + mid] > stripCount[threadIdx.x] ) 
            {
                stripCount[threadIdx.x] = stripCount[threadIdx.x + mid];
            }
        }
        __syncthreads();
        total = mid;
    } 

    if (threadIdx.x == 0)
    {
        atomicMax( (dir == DirTransposed ? &trMaxStripRowCountDev : &maxStripRowCountDev), stripCount[0]);
    }
}

template <_ImageDirection dir>
__global__ void convertScaleAbsKernel(gpu::PtrStepb debugSobelEqPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int width = depthWidth<dir>();
    int height = depthHeight<dir>();

    if (x < width && y < height)
    {
        float sobel = dirTex2D(dir, texSobel, texTrSobel, x, y);

        uchar res = (uchar)(fabsf(sobel) / (float)(_maxHistogramSizeDev[0]) * 255.0f + 0.5f);
        *(debugSobelEqPtr.ptr(y) + x) = res;
    }
}

template <_ImageDirection dir>
__global__ void refineDebugImageKernel(gpu::PtrStepSzb debugPtr, gpu::PtrStepb sobelEqPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int width = depthWidth<dir>();
    int height = depthHeight<dir>();

    if (x < width && y < height)
    {
        uchar* dstPixel = debugPtr.ptr(y) + x * 3;

        if (dstPixel[0] == 255 || dstPixel[1] == 255 || dstPixel[2] == 255)
		{
			//leave as is
		} 
		else
		{
            uchar sobelEq = *(sobelEqPtr.ptr(y) + x);
            float sobelVal = dirTex2D(dir, texSobel, texTrSobel, x, y);

            if (sobelVal >= 0)
            {
                dstPixel[0] = 0;
				dstPixel[2] = sobelEq;
            } else 
            {
                dstPixel[0] = sobelEq;
				dstPixel[2] = 0;
            }
            dstPixel[1] = 0;
		}
    }
}


__global__ void applySkinColorModel(gpu::PtrStepf luvPtr, gpu::PtrStepf pdfPtr)
{
    const int nComp = 3;
    const float mu[nComp][2] = {{ 9.57907488860893f,          12.7703451268183f},
                                { 25.0879013587047f,          35.3988094238412f},
                                { 19.8826767803543f,          23.1472246974151f}};
    const float conv[nComp][2] = { { 47.8492848747446f,          79.3673906277784f},
                                    {  170.235884002246f,          156.732288245996f},
                                    {  56.1710526374039f,          73.2031869267223f} };
    const float prop[nComp] = { 0.328943172607604f,         0.271925127240458f,         0.399131700151939f};

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < _settingsDev[0].rgbWidth && y < _settingsDev[0].rgbHeight)
    {
        float* luv = luvPtr.ptr(y) + x * 3;
        float u = luv[1];
        float v = luv[2];

        float p = 0;
        
        #pragma unroll
        for (int i = 0; i < nComp; i++)
        {
            float d = 1.f / (2 * HIP_PI_F * sqrt(conv[i][0] * conv[i][1]));
            float e = expf(-0.5f * (powf(u - mu[i][0], 2) / conv[i][0] + powf(v - mu[i][1], 2) / conv[i][1]));
            p += prop[i] * d * e;
        }

        float* dst = pdfPtr.ptr(y) + x;
        //*dst = p * 500.f;
        //*dst = luv[0] / 100.0f;
        //*dst = p * 1000.f;
        *dst = p;
    }
}

void Detector::gpuProcess()
{
    hipStream_t cudaStreamDepthDebug = gpu::StreamAccessor::getStream(_gpuStreamDepthDebug);
    hipStream_t cudaStreamDepthWorking = gpu::StreamAccessor::getStream(_gpuStreamDepthWorking);
    hipStream_t cudaStreamTransposedDepthDebug = gpu::StreamAccessor::getStream(_gpuStreamTransposedDepthDebug);
    hipStream_t cudaStreamTransposedDepthWorking = gpu::StreamAccessor::getStream(_gpuStreamTransposedDepthWorking);
    hipStream_t cudaStreamRgbWorking = gpu::StreamAccessor::getStream(_gpuStreamRgbWorking);
    
    _depthFrameGpu.upload(_depthFrame);
    _transposedDepthFrameGpu.upload(_transposedDepthFrame);
    //_gpuStreamDepthWorking.enqueueUpload(_depthFrame, _depthFrameGpu);
    
    //Looks like when running Sobel async, visual profiler won't generate any timeline.
    cv::gpu::Sobel(_depthFrameGpu, _sobelFrameGpu, CV_32F, 1, 0, _sobelFrameBufferGpu, 5, -1);
    //cv::gpu::Sobel(_depthFrameGpu, _sobelFrameGpu, CV_32F, 1, 0, _sobelFrameBufferGpu, 5, -1.0f, BORDER_DEFAULT, -1, _gpuStreamDepthWorking);
    //_gpuStreamDepthWorking.waitForCompletion();

    cv::gpu::Sobel(_transposedDepthFrameGpu, _transposedSobelFrameGpu, CV_32F, 1, 0, _transposedSobelFrameBufferGpu, 5, -1);

    _gpuStreamDepthWorking.enqueueMemSet(_debugFrameGpu, Scalar(0,0,0));
    _gpuStreamTransposedDepthWorking.enqueueMemSet(_transposedDebugFrameGpu, Scalar(0,0,0));
    
    //bind sobel for future usage
    hipChannelFormatDesc descSobel = hipCreateChannelDesc<float>();
    gpu::PtrStepSzb ptrSobel(_sobelFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texSobel, ptrSobel.data, descSobel, ptrSobel.cols, ptrSobel.rows, ptrSobel.step));

    hipChannelFormatDesc descTrSobel = hipCreateChannelDesc<float>();
    gpu::PtrStepSzb ptrTrSobel(_transposedSobelFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texTrSobel, ptrTrSobel.data, descTrSobel, ptrTrSobel.cols, ptrTrSobel.rows, ptrTrSobel.step));

    //bind depth
    hipChannelFormatDesc descDepth = hipCreateChannelDesc<ushort>();
    gpu::PtrStepSzb ptrDepth(_depthFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texDepth, ptrDepth.data, descDepth, ptrDepth.cols, ptrDepth.rows, ptrDepth.step));
    
    hipChannelFormatDesc descTrDepth = hipCreateChannelDesc<ushort>();
    gpu::PtrStepSzb ptrTrDepth(_transposedDepthFrameGpu);
    cudaSafeCall(hipBindTexture2D(NULL, texTrDepth, ptrTrDepth.data, descTrDepth, ptrTrDepth.cols, ptrTrDepth.rows, ptrTrDepth.step));


    //find strips on stream2: upload data
    //TODO: what if maximum thread < depthHeight? 
    //the third params: shared memory size in BYTES
    int* maxStripRowCountDevPtr;
    cudaSafeCall(hipGetSymbolAddress((void**)&maxStripRowCountDevPtr, maxStripRowCountDev));
    cudaSafeCall(hipMemsetAsync(maxStripRowCountDevPtr, 0, sizeof(int), cudaStreamDepthWorking));

    int* trMaxStripRowCountDevPtr;
    cudaSafeCall(hipGetSymbolAddress((void**)&trMaxStripRowCountDevPtr, trMaxStripRowCountDev));
    cudaSafeCall(hipMemsetAsync(trMaxStripRowCountDevPtr, 0, sizeof(int), cudaStreamTransposedDepthWorking));

    //find strips on stream 2: kernel call
    //turns out 1 block is the best even though profiler suggests more blocks
    int stripThread = _settings.depthHeight;    
    int stripBlock = 1; //divUp(_settings.depthHeight, nThread);
    findStripsKernel<DirDefault><<<stripBlock, stripThread, stripThread * sizeof(int), cudaStreamDepthWorking>>>(_debugFrameGpu, _stripsDev);
    
    int trStripThread = _settings.depthWidth;    
    int trStripBlock = 1; //divUp(_settings.depthHeight, nThread);
    findStripsKernel<DirTransposed><<<trStripBlock, trStripThread, trStripThread * sizeof(int), cudaStreamTransposedDepthWorking>>>
        (_transposedDebugFrameGpu, _transposedStripsDev);
    

    //rgb manipulation
    _gpuStreamRgbWorking.enqueueUpload(_rgbFrame, _rgbFrameGpu);
    _gpuStreamRgbWorking.enqueueConvert(_rgbFrameGpu, _rgbLabFrameGpu, CV_32FC3, 1.f / 255.f);
    gpu::cvtColor(_rgbLabFrameGpu, _rgbLabFrameGpu, CV_RGB2Luv, 0, _gpuStreamRgbWorking);

    //rgb skin color model
    dim3 rgbThreads(16, 32);
    dim3 rgbGrid(divUp(_settings.rgbWidth, rgbThreads.x), divUp(_settings.rgbHeight, rgbThreads.y));
    applySkinColorModel<<<rgbGrid, rgbThreads, 0, cudaStreamRgbWorking>>>(_rgbLabFrameGpu, _rgbPdfFrameGpu);
    cudaSafeCall(hipGetLastError());

    //refine debug image
    dim3 threads(16, 32);
    dim3 grid(divUp(_settings.depthWidth, threads.x), divUp(_settings.depthHeight, threads.y));
    if (DRAW_DEBUG_IMAGE)
    {
        convertScaleAbsKernel<DirDefault><<<grid, threads, 0, cudaStreamDepthDebug>>>(_debugSobelEqFrameGpu);
        gpu::equalizeHist(_debugSobelEqFrameGpu, _debugSobelEqFrameGpu, _debugSobelEqHistGpu, _debugSobelEqBufferGpu, _gpuStreamDepthDebug);
    }

    //refine transposed debug image
    dim3 trThreads(16, 32);
    dim3 trGrid(divUp(_settings.depthHeight, threads.x), divUp(_settings.depthWidth, threads.y));
    
    if (DRAW_DEBUG_IMAGE)
    {
        convertScaleAbsKernel<DirTransposed><<<trGrid, trThreads, 0, cudaStreamTransposedDepthDebug>>>(_transposedDebugSobelEqFrameGpu);
        gpu::equalizeHist(_transposedDebugSobelEqFrameGpu, _transposedDebugSobelEqFrameGpu, _transposedDebugSobelEqHistGpu, 
            _transposedDebugSobelEqBufferGpu, _gpuStreamTransposedDepthDebug);
    }

    //rgb download
    _gpuStreamRgbWorking.enqueueDownload(_rgbPdfFrameGpu, _rgbPdfFrame);

    //find strips: download data
    cudaSafeCall(hipMemcpyFromSymbolAsync(&_maxStripRowCount, HIP_SYMBOL(maxStripRowCountDev), sizeof(int), 0, hipMemcpyDeviceToHost, cudaStreamDepthWorking));
    //download strips
    //download effective data, there are maxStripCount + 1 rows. The extra row stores count of strips for each column
    cudaSafeCall(hipMemcpyAsync(_stripsHost, _stripsDev, _maxStripRowCount * _settings.depthHeight * sizeof(_OmniTouchStripDev), 
        hipMemcpyDeviceToHost, cudaStreamDepthWorking));
    //TODO: according to profiler, this trick seems not necessary. consider optimize for coelesence? 
  
    cudaSafeCall(hipMemcpyFromSymbolAsync(&_transposedMaxStripRowCount, HIP_SYMBOL(trMaxStripRowCountDev), sizeof(int), 0, 
        hipMemcpyDeviceToHost, cudaStreamTransposedDepthWorking));
    cudaSafeCall(hipMemcpyAsync(_transposedStripsHost, _transposedStripsDev, _transposedMaxStripRowCount * _settings.depthWidth * sizeof(_OmniTouchStripDev), 
        hipMemcpyDeviceToHost, cudaStreamDepthWorking));
    
    if (DRAW_DEBUG_IMAGE)
    {
        _gpuStreamDepthDebug.waitForCompletion();
        _gpuStreamTransposedDepthDebug.waitForCompletion();
    }
    
    _gpuStreamDepthWorking.waitForCompletion();  
    _gpuStreamTransposedDepthWorking.waitForCompletion();
    cudaSafeCall(hipGetLastError());

    //draw the debug image
    if (DRAW_DEBUG_IMAGE)
    {
        refineDebugImageKernel<DirDefault><<<grid, threads, 0, cudaStreamDepthDebug>>>(_debugFrameGpu, _debugSobelEqFrameGpu);
        refineDebugImageKernel<DirTransposed><<<trGrid, trThreads, 0, cudaStreamTransposedDepthDebug>>>(_transposedDebugFrameGpu, _transposedDebugSobelEqFrameGpu);
    }

    _gpuStreamDepthDebug.enqueueDownload(_debugFrameGpu, _debugFrame);
    _gpuStreamTransposedDepthDebug.enqueueDownload(_transposedDebugFrameGpu, _transposedDebugFrame);
    _gpuStreamDepthDebug.waitForCompletion();
    _gpuStreamTransposedDepthDebug.waitForCompletion();
    cudaSafeCall(hipGetLastError());    
               
    //unbind textures
    cudaSafeCall(hipUnbindTexture(texSobel));
    cudaSafeCall(hipUnbindTexture(texDepth));
    cudaSafeCall(hipUnbindTexture(texTrSobel));
    cudaSafeCall(hipUnbindTexture(texTrDepth));
    
    _gpuStreamRgbWorking.waitForCompletion();
    
}

